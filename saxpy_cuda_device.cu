#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

extern "C"
__device__ 
float saxpy_dev(float a, float x, float y)
{
  return a * x + y;
}

extern "C"
__device__
float foo(float in, float multiplier)
{
    __half in_half = __float2half(in);
    __half multiplier_half = __float2half(multiplier);
    __half out_half =  __hmul(in_half, multiplier_half);
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);   
    return __half2float(out_half);
}
