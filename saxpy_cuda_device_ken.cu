#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
extern "C"
__device__ 
float saxpy_dev(float a, float x, float y)
{
  return a * x + y;
}
extern "C"
__device__
__half saxpy_dev_half(__half a, __half x, __half y)
{
  return a * x + y;
}
